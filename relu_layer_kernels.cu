#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_runtime.h"

extern "C" {
  #include "relu_layer_kernels.h"
}

__global__ void relu_kernel(float *in, float *out) {
  const int col = blockIdx.x * blockDim.x + threadIdx.x;
  float v = in[col];
  if (v < 0) v = 0;
  out[col] = v;
}

void activate_relu_gpu(float *in, float *out) {
  // 24 x 24 * 8 => 24 * 24 * 8
  float *device_in;
  float *device_out;

  hipMalloc((void **) &device_in, 8 * 24 * 24 * sizeof(float));
  hipMalloc((void **) &device_out, 8 * 24 * 24 * sizeof(float));

  hipMemcpy(device_in, in, 24 * 24 * 8 * sizeof(float), hipMemcpyHostToDevice);

  relu_kernel<<<24, 24 * 8>>>(device_in, device_out);
  hipDeviceSynchronize();

  hipMemcpy(out, device_out, sizeof(float) * 8 * 24 * 24, hipMemcpyDeviceToHost);
}
